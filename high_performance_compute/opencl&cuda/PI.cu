#include "hip/hip_runtime.h"
// cudaPi.cpp : Defines the entry point for the console application.
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <tchar.h>
#define NUM_THREAD 1024
#define NUM_BLOCK 1
__global__ void cal_pi(double *sum, long long nbin, float step, long long nthreads, long long nblocks) {
    long long i;
    float x;
    long long idx = blockIdx.x*blockDim.x+threadIdx.x;
    for (i=idx; i< nbin; i+=nthreads*nblocks) {
        x = (i+0.5)*step;
        sum[idx] = sum[idx]+4.0/(1.+x*x);
    }
}
int _tmain(int argc, _TCHAR* argv[]) {
    long long tid;
    double pi = 0;
    long long num_steps = 100000000;
    float step = 1./(float)num_steps;
    long long size = NUM_THREAD*NUM_BLOCK*sizeof(double);
    clock_t before, after;
    double *sumHost, *sumDev;
    sumHost = (double *)malloc(size);
    hipMalloc((void **)&sumDev, size);// Initialize array in device to 0
    hipMemset(sumDev, 0, size);
    before = clock();// Do calculation on device
    printf("Before Compute \n\n");
    dim3 numBlocks(NUM_BLOCK,1,1);
    dim3 threadsPerBlock(NUM_THREAD,1,1);
    cal_pi <<<numBlocks, threadsPerBlock>>> (sumDev, (int)num_steps, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
    printf("After Compute \n\n");// Retrieve result from device and store it in host array
    hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
    printf("After Copy \n\n");
    for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++) {
        pi = pi+sumHost[tid];
    }
    pi = pi*step;
    after = clock();
    printf("The value of PI is %15.12f\n",pi);
    printf("The time to calculate PI was %f seconds\n",((float)(after - before)/1000.0));
    free(sumHost);
    hipFree(sumDev);
    return 0;
}