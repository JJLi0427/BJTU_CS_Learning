// add.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
// __global__ 变量声明符，作用是将add函数变成可以在GPU上运行的函数
// __global__ 函数被称为kernel，
// 在 GPU 上运行的代码通常称为设备代码（device code），而在 CPU 上运行的代码是主机代码（host code）。
__global__ 
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<25;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    // 内存分配，在GPU或者CPU上统一分配内存
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
}

    // Run kernel on 1M elements on the GPU
    // execution configuration, 执行配置
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    // CPU需要等待cuda上的代码运行完毕，才能对数据进行读取
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}